#include "hip/hip_runtime.h"
#include "test_common.h"
#include "AriesEngine/cpu_algorithm.h"
#include "CudaAcc/AriesSqlOperator.h"
#include "CudaAcc/AriesEngineAlgorithm.h"
using namespace aries_acc;
static const char* DB_NAME = "scale_1";

class GPUTimer {
public:
    hipEvent_t start, stop;

    GPUTimer() {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }
    virtual ~GPUTimer() { }

    template <typename Func>
    float timing(Func func) {
        float perf;

        hipEventRecord(start);

        func();

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipDeviceSynchronize();

        hipEventElapsedTime(&perf, start, stop);

        return perf;
    }
};

__global__ void calc_measure_sig_add( const int8_t* input_a, const int8_t* input_b, const int8_t* input_c, const int8_t* input_d, size_t tupleNum, size_t item_size, char *output )
{
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for( int64_t i = tid; i < tupleNum; i += stride )
    {
        aries_acc::Decimal columnId_1_( (CompactDecimal*)(input_a+i*item_size), 12, 2);
        aries_acc::Decimal columnId_2_( (CompactDecimal*)(input_b+i*item_size), 12, 2);
        aries_acc::Decimal columnId_3_( (CompactDecimal*)(input_c+i*item_size), 12, 2);
        aries_acc::Decimal columnId_4_( (CompactDecimal*)(input_d+i*item_size), 12, 2);
        aries_acc::Decimal columnId_5_ = columnId_1_ + columnId_2_ + columnId_3_ + columnId_4_;
        auto tmp = output + i * (size_t)7;
        columnId_5_.ToCompactDecimal(tmp, 7);
    }
}

__global__ void calc_measure_mlt_add( const int8_t* input_a, const int8_t* input_b, const int8_t* input_c, const int8_t* input_d, size_t tupleNum, size_t item_size, char *output )
{
    int32_t group_thread=threadIdx.x & TPI-1;
    int32_t index = ((long long)blockIdx.x*blockDim.x + threadIdx.x)/TPI;
    if(index>=tupleNum)
        return;
    uint32_t var_1[LIMBS] = {0};
    uint8_t var_1_sign = 0;
    uint32_t var_2[LIMBS] = {0};
    uint8_t var_2_sign = 0;
    uint32_t var_3[LIMBS] = {0};
    uint8_t var_3_sign = 0;
    uint32_t var_4[LIMBS] = {0};
    uint8_t var_4_sign = 0;

    char *var_1_temp = (char *)(input_a+index*item_size);
    var_1_temp += 5;
    char c_1= *var_1_temp;
    var_1_sign = GET_SIGN_FROM_BIT(c_1);
    if(group_thread < 0){
            aries_memcpy(var_1, ((CompactDecimal*)( input_a+index*item_size )) + group_thread * 64, 64);
    }
    if(group_thread == 0){
            aries_memcpy(var_1, ((CompactDecimal*)( input_a+index*item_size )) + group_thread * 64, 6);
            char *inner_temp = (char *)(var_1);
            inner_temp += 6 - 1;
            *inner_temp = *inner_temp & 0x7f;
    }

    char *var_2_temp = (char *)( input_b+index*item_size );
    var_2_temp += 5;
    char c_2= *var_2_temp;
    var_2_sign = GET_SIGN_FROM_BIT(c_2);
    if(group_thread < 0){
            aries_memcpy(var_2, ((CompactDecimal*)( input_b+index*item_size )) + group_thread * 64, 64);
    }
    if(group_thread == 0){
            aries_memcpy(var_2, ((CompactDecimal*)( input_b+index*item_size )) + group_thread * 64, 6);
            char *inner_temp = (char *)(var_2);
            inner_temp += 6 - 1;
            *inner_temp = *inner_temp & 0x7f;
    }

    uint32_t var_0[LIMBS] = {0};
    uint8_t var_0_sign = 0;
    var_0_sign = aries_acc::operator_add(var_0, var_1, var_2, 0, var_1_sign, var_2_sign);

    char *var_3_temp = (char *)( input_c+index*item_size );
    var_3_temp += 5;
    char c_3= *var_3_temp;
    var_3_sign = GET_SIGN_FROM_BIT(c_3);
    if(group_thread < 0){
            aries_memcpy(var_3, ((CompactDecimal*)( input_c+index*item_size )) + group_thread * 64, 64);
    }
    if(group_thread == 0){
            aries_memcpy(var_3, ((CompactDecimal*)( input_c+index*item_size )) + group_thread * 64, 6);
            char *inner_temp = (char *)(var_3);
            inner_temp += 6 - 1;
            *inner_temp = *inner_temp & 0x7f;
    }

    var_0_sign = aries_acc::operator_add(var_0, var_0, var_3, 0, var_0_sign, var_3_sign);

    char *var_4_temp = (char *)( input_d+index*item_size );
    var_4_temp += 5;
    char c_4= *var_4_temp;
    var_4_sign = GET_SIGN_FROM_BIT(c_4);
    if(group_thread < 0){
            aries_memcpy(var_4, ((CompactDecimal*)( input_d+index*item_size )) + group_thread * 64, 64);
    }
    if(group_thread == 0){
            aries_memcpy(var_4, ((CompactDecimal*)( input_d+index*item_size )) + group_thread * 64, 6);
            char *inner_temp = (char *)(var_4);
            inner_temp += 6 - 1;
            *inner_temp = *inner_temp & 0x7f;
    }
    
    var_0_sign = aries_acc::operator_add(var_0, var_0, var_4, 0, var_0_sign, var_4_sign);
    auto ans_tmp = output +  (long long)(index) * 7 + group_thread * 64;
    if(group_thread < 0){
            aries_memcpy(ans_tmp, var_0, 64);
    }
    if(group_thread == 0){
            aries_memcpy(ans_tmp, var_0, 7);
    }
    if(group_thread==0){
            char *buf = output + (long long)(index) * 7;
            SET_SIGN_BIT( buf[7-1], var_0_sign);
    }
}

__global__ void mem_access_mlt_add( const int8_t* input_a, const int8_t* input_b, const int8_t* input_c, const int8_t* input_d, size_t tupleNum, size_t item_size, char *output )
{
    int32_t group_thread=threadIdx.x & TPI-1;
    int32_t index = ((long long)blockIdx.x*blockDim.x + threadIdx.x)/TPI;
    if(index>=tupleNum)
        return;
    uint32_t var_1[LIMBS] = {0};
    uint8_t var_1_sign = 0;
    uint32_t var_2[LIMBS] = {0};
    uint8_t var_2_sign = 0;
    uint32_t var_3[LIMBS] = {0};
    uint8_t var_3_sign = 0;
    uint32_t var_4[LIMBS] = {0};
    uint8_t var_4_sign = 0;

    char *var_1_temp = (char *)(input_a+index*item_size);
    var_1_temp += 5;
    char c_1= *var_1_temp;
    var_1_sign = GET_SIGN_FROM_BIT(c_1);
    if(group_thread < 0){
            aries_memcpy(var_1, ((CompactDecimal*)( input_a+index*item_size )) + group_thread * 64, 64);
    }
    if(group_thread == 0){
            aries_memcpy(var_1, ((CompactDecimal*)( input_a+index*item_size )) + group_thread * 64, 6);
            char *inner_temp = (char *)(var_1);
            inner_temp += 6 - 1;
            *inner_temp = *inner_temp & 0x7f;
    }

    char *var_2_temp = (char *)( input_b+index*item_size );
    var_2_temp += 5;
    char c_2= *var_2_temp;
    var_2_sign = GET_SIGN_FROM_BIT(c_2);
    if(group_thread < 0){
            aries_memcpy(var_2, ((CompactDecimal*)( input_b+index*item_size )) + group_thread * 64, 64);
    }
    if(group_thread == 0){
            aries_memcpy(var_2, ((CompactDecimal*)( input_b+index*item_size )) + group_thread * 64, 6);
            char *inner_temp = (char *)(var_2);
            inner_temp += 6 - 1;
            *inner_temp = *inner_temp & 0x7f;
    }

    uint32_t var_0[LIMBS] = {0};
    uint8_t var_0_sign = 0;
    var_0_sign = aries_acc::operator_add(var_0, var_1, var_2, 0, var_1_sign, var_2_sign);

    char *var_3_temp = (char *)( input_c+index*item_size );
    var_3_temp += 5;
    char c_3= *var_3_temp;
    var_3_sign = GET_SIGN_FROM_BIT(c_3);
    if(group_thread < 0){
            aries_memcpy(var_3, ((CompactDecimal*)( input_c+index*item_size )) + group_thread * 64, 64);
    }
    if(group_thread == 0){
            aries_memcpy(var_3, ((CompactDecimal*)( input_c+index*item_size )) + group_thread * 64, 6);
            char *inner_temp = (char *)(var_3);
            inner_temp += 6 - 1;
            *inner_temp = *inner_temp & 0x7f;
    }

    var_0_sign = aries_acc::operator_add(var_0, var_0, var_3, 0, var_0_sign, var_3_sign);

    char *var_4_temp = (char *)( input_d+index*item_size );
    var_4_temp += 5;
    char c_4= *var_4_temp;
    var_4_sign = GET_SIGN_FROM_BIT(c_4);
    if(group_thread < 0){
            aries_memcpy(var_4, ((CompactDecimal*)( input_d+index*item_size )) + group_thread * 64, 64);
    }
    if(group_thread == 0){
            aries_memcpy(var_4, ((CompactDecimal*)( input_d+index*item_size )) + group_thread * 64, 6);
            char *inner_temp = (char *)(var_4);
            inner_temp += 6 - 1;
            *inner_temp = *inner_temp & 0x7f;
    }
    
    var_0_sign = aries_acc::operator_add(var_0, var_0, var_4, 0, var_0_sign, var_4_sign);
    auto ans_tmp = output +  (long long)(index) * 7 + group_thread * 64;
    if(group_thread < 0){
            aries_memcpy(ans_tmp, var_0, 64);
    }
    if(group_thread == 0){
            aries_memcpy(ans_tmp, var_0, 7);
    }
    if(group_thread==0){
            char *buf = output + (long long)(index) * 7;
            SET_SIGN_BIT( buf[7-1], var_0_sign);
    }
}

__global__ void mem_access_mlt_add_recycle( const int8_t* input_a, const int8_t* input_b, size_t tupleNum, size_t item_size, char *output )
{
    int32_t group_thread=threadIdx.x & TPI-1;
    int32_t index = ((long long)blockIdx.x*blockDim.x + threadIdx.x)/TPI;
    if(index>=tupleNum)
        return;
    uint32_t var_1[LIMBS] = {0};
    uint8_t var_1_sign = 0;
    uint32_t var_2[LIMBS] = {0};
    uint8_t var_2_sign = 0;
    uint32_t var_3[LIMBS] = {0};
    uint8_t var_3_sign = 0;
    uint32_t var_4[LIMBS] = {0};
    uint8_t var_4_sign = 0;

    char *var_1_temp = (char *)(input_a+index*item_size);
    var_1_temp += 5;
    char c_1= *var_1_temp;
    var_1_sign = GET_SIGN_FROM_BIT(c_1);
    if(group_thread == 0){
            aries_memcpy(var_1, ((CompactDecimal*)( input_a+index*item_size )) + group_thread * LIMBS * 4, 6);
            char *inner_temp = (char *)(var_1);
            inner_temp += 6 - 1;
            *inner_temp = *inner_temp & 0x7f;
            aries_memcpy(var_3, var_1, 6);
    }

    char *var_2_temp = (char *)( input_b+index*item_size );
    var_2_temp += 5;
    char c_2= *var_2_temp;
    var_2_sign = GET_SIGN_FROM_BIT(c_2);
    if(group_thread == 0){
            aries_memcpy(var_2, ((CompactDecimal*)( input_b+index*item_size )) + group_thread * LIMBS * 4, 6);
            char *inner_temp = (char *)(var_2);
            inner_temp += 6 - 1;
            *inner_temp = *inner_temp & 0x7f;
            aries_memcpy(var_4, var_2, 6);
    }

    var_3_sign = var_1_sign;
    var_4_sign = var_2_sign;
//     for (int i = 0; i < LIMBS; i++){
//         var_3[i] = var_1[i];
//         var_4[i] = var_2[i];
//     }
    

    uint32_t var_0[LIMBS] = {0};
    uint8_t var_0_sign = 0;
    var_0_sign = aries_acc::operator_add(var_0, var_1, var_2, 0, var_1_sign, var_2_sign);
    var_0_sign = aries_acc::operator_add(var_0, var_0, var_3, 0, var_0_sign, var_3_sign);
    var_0_sign = aries_acc::operator_add(var_0, var_0, var_4, 0, var_0_sign, var_4_sign);
    auto ans_tmp = output +  (long long)(index) * 7 + group_thread * LIMBS * 4;
    if(group_thread==0){
        aries_memcpy(ans_tmp, var_0, 7);
        char *buf = output + (long long)(index) * 7;
        SET_SIGN_BIT( buf[7-1], var_0_sign);
    }
}

__global__ void mem_access_sig_add( const int8_t* input_a, const int8_t* input_b, size_t tupleNum, size_t item_size, char *output )
{
    int stride = blockDim.x * gridDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for( int64_t i = tid; i < tupleNum; i += stride )
    {
        aries_acc::Decimal columnId_1_( (CompactDecimal*)(input_a+i*item_size), 12, 2);
        aries_acc::Decimal columnId_2_( (CompactDecimal*)(input_b+i*item_size), 12, 2);
        aries_acc::Decimal columnId_3_ = columnId_1_ + columnId_2_ + columnId_1_ + columnId_2_;
        auto tmp = output + i * (size_t)7;
        columnId_3_.ToCompactDecimal(tmp, 7);
    }
}

void main_execute_calc_measure(int type){
        GPUTimer gpuTimer;
        float gpuPerf = 0.0;
        // 从 data 目录中读取数据 这里读取到的数据是 compactDecimal
        // 这里取到了两组数据 lineitem 的第五列 和 第七列 它们的格式都是 prec = 12 frac = 2
        standard_context_t context;
        AriesDataBufferSPtr l_quantity = ReadColumn( DB_NAME, "lineitem", 5 );
        AriesDataBufferSPtr l_extendedprice = ReadColumn( DB_NAME, "lineitem", 6 );
        AriesDataBufferSPtr l_discount = ReadColumn( DB_NAME, "lineitem", 7 );
        AriesDataBufferSPtr l_tax = ReadColumn( DB_NAME, "lineitem", 8 );

        // 两组数据的列数 和 两组数据 compactDecimal的字节数
        size_t calc_number = l_quantity->GetItemCount();
        size_t byte_per_op_unit = GetDecimalRealBytes(12, 2);

        cout<< byte_per_op_unit <<endl;
        
        int8_t *c_q_gpu, *c_e_gpu, *c_d_gpu, *c_t_gpu;
        hipMalloc((void **)&c_q_gpu, calc_number * byte_per_op_unit);
        hipMalloc((void **)&c_e_gpu, calc_number * byte_per_op_unit);
        hipMalloc((void **)&c_d_gpu, calc_number * byte_per_op_unit);
        hipMalloc((void **)&c_t_gpu, calc_number * byte_per_op_unit);
        // 将数据拷贝到 gpu 上
        hipMemcpy(c_q_gpu, l_quantity->GetData(), calc_number * byte_per_op_unit, hipMemcpyHostToDevice);
        hipMemcpy(c_e_gpu, l_extendedprice->GetData(), calc_number * byte_per_op_unit, hipMemcpyHostToDevice);
        hipMemcpy(c_d_gpu, l_discount->GetData(), calc_number * byte_per_op_unit, hipMemcpyHostToDevice);
        hipMemcpy(c_t_gpu, l_tax->GetData(), calc_number * byte_per_op_unit, hipMemcpyHostToDevice);

        char *ans_cpu, *ans_gpu;
        // 结果需要的内存空间的大小
        size_t ans_sum_byte;
        // 输出的文件
        FILE *fp;
        if(type == 1 || type == 2){
                // 计算 l_quantity + l_discount 列 它们的结果类型 ans.prec = 13 ans.frac = 2
                int byte_per_ans_unit = GetDecimalRealBytes(15, 2);
                ans_sum_byte = byte_per_ans_unit * calc_number;

                cout<< byte_per_ans_unit <<endl;

                // 申请结果在 CPU 和 GPU 上的空间
                ans_cpu = (char *)malloc(ans_sum_byte);
                hipMalloc((void **)&ans_gpu, ans_sum_byte);

                if(type == 1){
                        // 调用 kernel
                        int threadN = 256;
                        size_t blockN = (calc_number - 1)/threadN + 1;
                        gpuPerf = gpuTimer.timing( [&](){
                                calc_measure_sig_add<<<blockN, threadN>>>(c_q_gpu, c_e_gpu, c_d_gpu, c_t_gpu, calc_number, byte_per_op_unit, ans_gpu);
                        });
                        cout<<"Calc_measure --> sig_add :"<<gpuPerf<<"ms"<<endl;
                        fp = fopen("../Varify/Calc_measure_sig_add.txt", "ab+");
                }
                else{
                        // 调用 kernel
                        int threadN = 256;
                        size_t blockN = (calc_number*TPI - 1)/threadN + 1;
                        gpuPerf = gpuTimer.timing( [&](){
                                calc_measure_mlt_add<<<blockN, threadN>>>(c_q_gpu, c_e_gpu, c_d_gpu, c_t_gpu, calc_number, byte_per_op_unit, ans_gpu);
                        });
                        cout<<"Calc_measure --> mlt_add :"<<gpuPerf<<"ms"<<endl;
                        fp = fopen("../Varify/Calc_measure_mlt_add.txt", "ab+");
                }
        }

        
        hipMemcpy(ans_cpu, ans_gpu, ans_sum_byte, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        // compact 输出
        // fseek(fp, 0, SEEK_SET);
        // fwrite(ans_cpu ,1 , ans_sum_byte, fp);

        // // decimal 输出
        aries_acc::Decimal *ans_dec_cpu;
        ans_dec_cpu = (aries_acc::Decimal *)malloc(calc_number * sizeof(aries_acc::Decimal));
        for(size_t i=0; i<calc_number; i++){
                int byte_per_ans_unit = GetDecimalRealBytes(15, 2); 
                ans_dec_cpu[i] = aries_acc::Decimal((CompactDecimal *)(ans_cpu+i*byte_per_ans_unit), 15, 2);    
        }
        for(int i=0; i<calc_number; i++){
                char result[2048];
                for(int j=INDEX_LAST_DIG ; j>=0 ;j--){
                        sprintf(result+(INDEX_LAST_DIG-j)*8,"%08x",ans_dec_cpu[i].v[j]);
                }
                // sprintf(result+NUM_TOTAL_DIG*8," sign = %d , frac = %d",ans_dec_cpu[i].sign,ans_dec_cpu[i].frac);
                int len = strlen(result);
                fwrite(result,len,1,fp);
                fwrite("\r\n",1,2,fp);
        }

        fclose(fp);

        free(ans_cpu);
        hipFree(ans_gpu);
}

void main_execute_mem_access(int type){
        GPUTimer gpuTimer;
        float gpuPerf = 0.0;
        // 从 data 目录中读取数据 这里读取到的数据是 compactDecimal
        // 这里取到了两组数据 lineitem 的第五列 和 第七列 它们的格式都是 prec = 12 frac = 2
        standard_context_t context;
        AriesDataBufferSPtr l_quantity = ReadColumn( DB_NAME, "lineitem", 5 );
        AriesDataBufferSPtr l_discount = ReadColumn( DB_NAME, "lineitem", 7 );

        // 两组数据的列数 和 两组数据 compactDecimal的字节数
        size_t calc_number = l_quantity->GetItemCount();
        size_t byte_per_op_unit = GetDecimalRealBytes(12, 2);

        cout<< byte_per_op_unit <<endl;
        
        int8_t *c_q_gpu, *c_d_gpu;
        hipMalloc((void **)&c_q_gpu, calc_number * byte_per_op_unit);
        hipMalloc((void **)&c_d_gpu, calc_number * byte_per_op_unit);
        // 将数据拷贝到 gpu 上
        hipMemcpy(c_q_gpu, l_quantity->GetData(), calc_number * byte_per_op_unit, hipMemcpyHostToDevice);
        hipMemcpy(c_d_gpu, l_discount->GetData(), calc_number * byte_per_op_unit, hipMemcpyHostToDevice);

        char *ans_cpu, *ans_gpu;
        // 结果需要的内存空间的大小
        size_t ans_sum_byte;
        // 输出的文件
        FILE *fp;
        if(type == 1 || type == 2 || type == 3){
                // 计算 l_quantity + l_discount 列 它们的结果类型 ans.prec = 13 ans.frac = 2
                int byte_per_ans_unit = GetDecimalRealBytes(15, 2);
                ans_sum_byte = byte_per_ans_unit * calc_number;

                cout<< byte_per_ans_unit <<endl;

                // 申请结果在 CPU 和 GPU 上的空间
                ans_cpu = (char *)malloc(ans_sum_byte);
                hipMalloc((void **)&ans_gpu, ans_sum_byte);

                if(type == 1){
                        // 调用 kernel
                        int threadN = 256;
                        size_t blockN = (calc_number - 1)/threadN + 1;
                        gpuPerf = gpuTimer.timing( [&](){
                                mem_access_sig_add<<<blockN, threadN>>>(c_q_gpu, c_d_gpu, calc_number, byte_per_op_unit, ans_gpu);
                        });
                        cout<<"Mem_access --> sig_add :"<<gpuPerf<<"ms"<<endl;
                        fp = fopen("../Varify/Mem_access_sig_add.txt", "ab+");
                }
                else if(type == 2){
                        int8_t *c_q_gpu_2, *c_d_gpu_2;
                        hipMalloc((void **)&c_q_gpu_2, calc_number * byte_per_op_unit);
                        hipMalloc((void **)&c_d_gpu_2, calc_number * byte_per_op_unit);
                        hipMemcpy(c_q_gpu_2, l_quantity->GetData(), calc_number * byte_per_op_unit, hipMemcpyHostToDevice);
                        hipMemcpy(c_d_gpu_2, l_discount->GetData(), calc_number * byte_per_op_unit, hipMemcpyHostToDevice);
                        // 调用 kernel
                        int threadN = 256;
                        size_t blockN = (calc_number*TPI - 1)/threadN + 1;
                        gpuPerf = gpuTimer.timing( [&](){
                                mem_access_mlt_add<<<blockN, threadN>>>(c_q_gpu, c_d_gpu, c_q_gpu_2, c_d_gpu_2, calc_number, byte_per_op_unit, ans_gpu);
                        });
                        cout<<"Mem_access --> mlt_add :"<<gpuPerf<<"ms"<<endl;
                        fp = fopen("../Varify/Mem_access_mlt_add.txt", "ab+");
                }
                else{
                        // 调用 kernel
                        int threadN = 256;
                        size_t blockN = (calc_number*TPI - 1)/threadN + 1;
                        gpuPerf = gpuTimer.timing( [&](){
                                mem_access_mlt_add_recycle<<<blockN, threadN>>>(c_q_gpu, c_d_gpu, calc_number, byte_per_op_unit, ans_gpu);
                        });
                        cout<<"Mem_access --> mlt_add_recycle :"<<gpuPerf<<"ms"<<endl;
                        fp = fopen("../Varify/Mem_access_mlt_add_recycle.txt", "ab+");
                }
        }

        
        hipMemcpy(ans_cpu, ans_gpu, ans_sum_byte, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        // compact 输出
        fseek(fp, 0, SEEK_SET);
        fwrite(ans_cpu ,1 , ans_sum_byte, fp);

        // // // decimal 输出
        // aries_acc::Decimal *ans_dec_cpu;
        // ans_dec_cpu = (aries_acc::Decimal *)malloc(calc_number * sizeof(aries_acc::Decimal));
        // for(size_t i=0; i<calc_number; i++){
        //         int byte_per_ans_unit = GetDecimalRealBytes(15, 2); 
        //         ans_dec_cpu[i] = aries_acc::Decimal((CompactDecimal *)(ans_cpu+i*byte_per_ans_unit), 15, 2);    
        // }
        // for(int i=0; i<calc_number; i++){
        //         char result[2048];
        //         for(int j=INDEX_LAST_DIG ; j>=0 ;j--){
        //                 sprintf(result+(INDEX_LAST_DIG-j)*8,"%08x",ans_dec_cpu[i].v[j]);
        //         }
        //         // sprintf(result+NUM_TOTAL_DIG*8," sign = %d , frac = %d",ans_dec_cpu[i].sign,ans_dec_cpu[i].frac);
        //         int len = strlen(result);
        //         fwrite(result,len,1,fp);
        //         fwrite("\r\n",1,2,fp);
        // }

        fclose(fp);

        free(ans_cpu);
        hipFree(ans_gpu);
}

TEST(Calc_measure, sig_add)
{
        main_execute_calc_measure(1);
}

TEST(Calc_measure, mlt_add)
{
        main_execute_calc_measure(2);
}

TEST(Mem_access_mesure, sig_add)
{
        main_execute_mem_access(1);
}

TEST(Mem_access_mesure, mlt_add)
{
        main_execute_mem_access(2);
}

TEST(Mem_access_mesure, mlt_add_recycle)
{
        main_execute_mem_access(3);
}