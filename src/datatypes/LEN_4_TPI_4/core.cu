#include "hip/hip_runtime.h"
#include "static_divide.cu"
#include "mp.cu"
#include "math.h"

namespace cgbn_LEN_4_TPI_4{

  class core {

      static const uint32_t        TPI_ONES=(1ull<<TPI_ONE)-1;

      public:

          __device__ __forceinline__ static uint32_t instance_sync_mask() {
              uint32_t group_thread=threadIdx.x & TPI_ONE-1, warp_thread=threadIdx.x & warpSize-1;
              
              return TPI_ONES<<(group_thread ^ warp_thread);
          }
          __device__ __forceinline__ static uint32_t sync_mask() {
              return instance_sync_mask();
          }

          __device__ __forceinline__ static uint32_t clz(const uint32_t a[LIMBS_ONE]) {
            uint32_t sync=sync_mask(), group_thread=threadIdx.x & TPI_ONE-1, warp_thread=threadIdx.x & warpSize-1;
            uint32_t clz, topclz;
            
            clz=cgbn_LEN_4_TPI_4::mpclz(a);
            topclz=__ballot_sync(sync, clz!=32*LIMBS_ONE);
            if(TPI_ONE<warpSize)
              topclz=topclz<<(warpSize-TPI_ONE)-(warp_thread-group_thread);
            topclz=__clz(topclz);
            if(topclz>=TPI_ONE)
              return LIMBS_ONE*TPI_ONE*32;
            return __shfl_sync(sync, (TPI_ONE-1-group_thread)*32*LIMBS_ONE + clz, 31-topclz, TPI_ONE)-LIMBS_ONE*TPI_ONE*32+LIMBS_ONE*TPI_ONE*32;
        }

        __device__ __forceinline__ static void rotate_right(const uint32_t sync, uint32_t r[], const uint32_t x[], const uint32_t numbits) {
          uint32_t rotate_bits=numbits & 0x1F, numlimbs=numbits>>5, threads=static_divide_small(numlimbs);

          numlimbs=numlimbs-threads*LIMBS_ONE;
          if(numlimbs==0) {
            #pragma unroll
            for(int32_t index=0;index<LIMBS_ONE;index++)
              r[index]=__shfl_sync(sync, x[index], threadIdx.x+threads, TPI_ONE);
          }
          else {
            mprotate_right(r, x, numlimbs);
            #pragma unroll
            for(int32_t index=0;index<LIMBS_ONE;index++)
              r[index]=__shfl_sync(sync, r[index], threadIdx.x+threads+(LIMBS_ONE-index<=numlimbs), TPI_ONE);
          }

          if(rotate_bits>0) {
            uint32_t fill=__shfl_sync(sync, r[0], threadIdx.x+1, TPI_ONE);

            mpright(r, r, rotate_bits, fill);
          }
        }

        __device__ __forceinline__ static void rotate_left(const uint32_t sync, uint32_t r[], const uint32_t x[], const uint32_t numbits) {
          // printf("drotate_left _in\n");
          uint32_t rotate_bits=numbits & 0x1F, numlimbs=numbits>>5, threads=static_divide_small(numlimbs);

          numlimbs=numlimbs-threads*LIMBS_ONE;
          if(numlimbs==0) {
            #pragma unroll
            for(int32_t index=0;index<LIMBS_ONE;index++)
              r[index]=__shfl_sync(sync, x[index], threadIdx.x-threads, TPI_ONE);
          }
          else {
            mprotate_left(r, x, numlimbs);
            #pragma unroll
            for(int32_t index=0;index<LIMBS_ONE;index++)
              r[index]=__shfl_sync(sync, r[index], threadIdx.x-threads-(index<numlimbs), TPI_ONE);
          }

          if(rotate_bits>0) {
            uint32_t fill=__shfl_sync(sync, r[LIMBS_ONE-1], threadIdx.x-1, TPI_ONE);

            mpleft(r, r, rotate_bits, fill);
          }
        }

        __device__ __forceinline__ static void bitwise_xor(uint32_t r[LIMBS_ONE], const uint32_t a[LIMBS_ONE], const uint32_t b[LIMBS_ONE]) {
          #pragma unroll
          for(int32_t index=0;index<LIMBS_ONE;index++)
            r[index]=a[index] ^ b[index];
        }
        
        __device__ __forceinline__ static void bitwise_mask_and(uint32_t r[], const uint32_t a[], const int32_t numbits) {
          // printf("dmask_and \n");
          int32_t group_thread=threadIdx.x & TPI_ONE-1, group_base=group_thread*LIMBS_ONE;
          int32_t bits=TPI_ONE*LIMBS_ONE*32;
          
          if(numbits>=bits || numbits<=-bits) {
            #pragma unroll
            for(int32_t index=0;index<LIMBS_ONE;index++)
              r[index]=a[index];
          }
          else if(numbits>=0) {
            int32_t limb=(numbits>>5)-group_base;
            int32_t straddle=uleft_wrap(0xFFFFFFFF, 0, numbits);

            #pragma unroll
            for(int32_t index=0;index<LIMBS_ONE;index++) {
              if(limb<index)
                r[index]=0;
              else if(limb>index)
                r[index]=a[index];
              else
                r[index]=a[index] & straddle;
            }
          }
          else {
            int32_t limb=(numbits+bits>>5)-group_base;
            int32_t straddle=uleft_wrap(0, 0xFFFFFFFF, numbits);

            #pragma unroll
            for(int32_t index=0;index<LIMBS_ONE;index++) {
              if(limb<index)
                r[index]=a[index];
              else if(limb>index)
                r[index]=0;
              else
                r[index]=a[index] & straddle;
            }
          }
        }

        __device__ __forceinline__ static uint32_t clzt(const uint32_t a[LIMBS_ONE]) {
          // printf("clzt 1\n");
          uint32_t sync=sync_mask(), group_thread=threadIdx.x & TPI_ONE-1, warp_thread=threadIdx.x & warpSize-1;
          uint32_t lor, topclz;

          lor=mplor(a);
          topclz=__ballot_sync(sync, lor!=0);
          if(TPI_ONE<warpSize)
            topclz=topclz<<(warpSize-TPI_ONE)-(warp_thread-group_thread);
          topclz= __clz(topclz);
          return umin(topclz, TPI_ONE);
        }

        __device__ __forceinline__ static int32_t resolve_add_a(const int32_t carry, uint32_t &x) {
          // printf("resolve_add in 3\n");
          uint32_t sync=core::sync_mask(), group_thread=threadIdx.x & TPI_ONE-1;
          uint32_t lane=(group_thread==0) ? 0 : 1<<(threadIdx.x & warpSize-1);
          uint32_t g, p, c;
          uint64_t sum;
        
          c=__shfl_up_sync(sync, carry, 1, TPI_ONE);
          c=(group_thread==0) ? 0 : c;
          x=add_cc(x, c);
          c=addc(0, 0);

          g=__ballot_sync(sync, c==1);
          p=__ballot_sync(sync, x==0xFFFFFFFF && group_thread!=0);
        
          // wrap the carry around  
          sum=make_wide(g, g) + make_wide(g, g) + make_wide(p, p);
          c=lane&(p^sum);

          x=x+(c!=0);
          c=uright_wrap(sum>>32, 0, threadIdx.x - group_thread + TPI_ONE) & 0x01;
          return __shfl_sync(sync, carry+c, TPI_ONE-1, TPI_ONE);
        }

        __device__ __forceinline__ static int32_t resolve_add_b(const int32_t carry, uint32_t x[LIMBS_ONE]) {
          // printf("resolve_add in 4\n");
          uint32_t sync=core::sync_mask(), group_thread=threadIdx.x & TPI_ONE-1;
          uint32_t lane=(group_thread==0) ? 0 : 1<<(threadIdx.x & warpSize-1);
          uint32_t g, p, c, land;
          uint64_t sum;
          
          c=__shfl_up_sync(sync, carry, 1, TPI_ONE);
          c=(group_thread==0) ? 0 : c;
          x[0]=add_cc(x[0], c);
          #pragma unroll
          for(int32_t index=1;index<LIMBS_ONE;index++) 
            x[index]=addc_cc(x[index], 0);
          c=addc(0, 0);
        
          land=mpland(x);
          g=__ballot_sync(sync, c==1);
          p=__ballot_sync(sync, land==0xFFFFFFFF && group_thread!=0);
        
          sum=make_wide(g, g) + make_wide(g, g) + make_wide(p, p);
          c=lane&(p^sum);
      
          x[0]=add_cc(x[0], c!=0);
          #pragma unroll
          for(int32_t index=1;index<LIMBS_ONE;index++)
            x[index]=addc_cc(x[index], 0);
          
          c=uright_wrap(sum>>32, 0, threadIdx.x - group_thread + TPI_ONE) & 0x01;
          return __shfl_sync(sync, carry+c, TPI_ONE-1, TPI_ONE);
        }
  };

}
