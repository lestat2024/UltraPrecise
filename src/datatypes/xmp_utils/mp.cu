#include "hip/hip_runtime.h"
#include "asm.cu"
#include "shifter_t.cu"
namespace cgbn {

    /* asm routines */
    __device__ __forceinline__ uint32_t add_cc(uint32_t a, uint32_t b);
    __device__ __forceinline__ uint32_t addc_cc(uint32_t a, uint32_t b);
    __device__ __forceinline__ uint32_t addc(uint32_t a, uint32_t b);
    __device__ __forceinline__ uint32_t sub_cc(uint32_t a, uint32_t b);
    __device__ __forceinline__ uint32_t subc_cc(uint32_t a, uint32_t b);
    __device__ __forceinline__ uint32_t subc(uint32_t a, uint32_t b);

    #define CGBN_INF_CHAIN 0xFFFFFFFF

    /* classes */
    template<uint32_t length=CGBN_INF_CHAIN, bool carry_in=false, bool carry_out=false>
    class chain_t {
        public:
            uint32_t _position;

            __device__ __forceinline__ chain_t();
            __device__ __forceinline__ ~chain_t();
            __device__ __forceinline__ uint32_t add(uint32_t a, uint32_t b);
            __device__ __forceinline__ uint32_t sub(uint32_t a, uint32_t b);
            __device__ __forceinline__ uint32_t madlo(uint32_t a, uint32_t b, uint32_t c);
            __device__ __forceinline__ uint32_t madhi(uint32_t a, uint32_t b, uint32_t c);
    };

    __device__ __forceinline__ uint32_t mpclz(const uint32_t a[]) {
        uint32_t word=0, count=0;
        
        #pragma unroll
        for(int32_t index=LIMBS-1;index>=0;index--) {
            word=(word!=0) ? word : a[index];
            count=(word!=0) ? count : (LIMBS-index)*32;
        }
        if(word!=0)
            count=count+__clz(word);
        return count;
    }

    __device__ __forceinline__ void mpleft(uint32_t r[], const uint32_t a[], const uint32_t numbits, const uint32_t fill=0) {
        #pragma unroll
        for(int32_t index=LIMBS-1;index>=1;index--)
            r[index]=uleft_clamp(a[index-1], a[index], numbits);
        r[0]=uleft_clamp(fill, a[0], numbits);
    }

    __device__ __forceinline__ uint32_t mpland(const uint32_t a[]) {
        uint32_t r=a[0];
        
        #pragma unroll
        for(int32_t index=1;index<LIMBS;index++)
        r=r & a[index];
        return r;
    }

    __device__ __forceinline__ uint32_t mplor(const uint32_t a[]) {
        uint32_t r=a[0];
        
        #pragma unroll
        for(int32_t index=1;index<LIMBS;index++)
        r=r | a[index];
        return r;
    }

    __device__ __forceinline__ void mpmul(uint32_t lo[], uint32_t hi[], const uint32_t a[], const uint32_t b[]) {
        uint32_t c;
        
        #pragma unroll
        for(int32_t index=0;index<LIMBS;index++) {
            lo[index]=0;
            hi[index]=0;
        }
        
        #pragma unroll
        for(int32_t i=0;i<LIMBS;i++) {
            chain_t<LIMBS,false,true> chain1;
            #pragma unroll
            for(int32_t j=0;j<LIMBS;j++) {
            if(i+j<LIMBS)
                lo[i+j]=chain1.madlo(a[i], b[j], lo[i+j]);
            else
                hi[i+j-LIMBS]=chain1.madlo(a[i], b[j], hi[i+j-LIMBS]);
            }
            if(i==0)
            c=0;
            else
            c=addc(0, 0);
            
            chain_t<LIMBS> chain2;
            #pragma unroll
            for(int32_t j=0;j<LIMBS-1;j++) {
            if(i+j+1<LIMBS)
                lo[i+j+1]=chain2.madhi(a[i], b[j], lo[i+j+1]);
            else
                hi[i+j+1-LIMBS]=chain2.madhi(a[i], b[j], hi[i+j+1-LIMBS]);
            }
            hi[i]=chain2.madhi(a[i], b[LIMBS-1], c);
        }
    }

    __device__ __forceinline__ void mpsub_cc(uint32_t r[], const uint32_t a[], const uint32_t b[]) {
        chain_t<LIMBS,false,true> chain;
        #pragma unroll
        for(int32_t index=0;index<LIMBS;index++)
            r[index]=chain.sub(a[index], b[index]);
    }

    __device__ __forceinline__ uint32_t mpsub(uint32_t r[], const uint32_t a[], const uint32_t b[]) {
        mpsub_cc(r, a, b);
        return subc(0, 0);
    }

    __device__ __forceinline__ uint32_t mpmul32(uint32_t r[], const uint32_t a[], const uint32_t b) {
        uint32_t carry=0;
        
        #pragma unroll
        for(int32_t index=0;index<LIMBS;index++) {
            uint32_t temp=a[index];
            
            r[index]=madlo_cc(temp, b, carry);
            carry=madhic(temp, b, 0);
        }
        return carry;
    }

    __device__ __forceinline__ void mpadd_cc(uint32_t r[], const uint32_t a[], const uint32_t b[]) {
        chain_t<LIMBS,false,true> chain;
        #pragma unroll
        for(int32_t index=0;index<LIMBS;index++)
            r[index]=chain.add(a[index], b[index]);
    }

    __device__ __forceinline__ uint32_t mpadd(uint32_t r[], const uint32_t a[], const uint32_t b[]) {
        mpadd_cc(r, a, b);
        return addc(0, 0);
    }

    __device__ __forceinline__ void mpsub32_cc(uint32_t r[], const uint32_t a[], const uint32_t b) {
        chain_t<LIMBS,false,true> chain;
        r[0]=chain.sub(a[0], b);
        #pragma unroll
        for(int32_t index=1;index<LIMBS;index++)
            r[index]=chain.sub(a[index], 0);
    }

    __device__ __forceinline__ uint32_t mpsub32(uint32_t r[], const uint32_t a[], const uint32_t b) {
        mpsub32_cc(r, a, b);
        return subc(0, 0);
    }

    __device__ __forceinline__ void mprotate_left(uint32_t r[], const uint32_t a[], const uint32_t numlimbs) {
        // printf("mprotate_left tid =%d max_roration = %d\n",threadIdx.x, MAX_ROTATION);
        #pragma unroll
        for(int32_t index=0;index<LIMBS;index++) 
            r[index]=a[index];
        
        if(LIMBS>bit_set<MAX_ROTATION>::high_bit*2)
            shifter_t<LIMBS, bit_set<MAX_ROTATION>::high_bit, true>::mprotate_left(r, numlimbs);
        else if((LIMBS-1&LIMBS)==0)
            shifter_t<LIMBS, LIMBS/2, false>::mprotate_left(r, numlimbs);
        else
            shifter_t<LIMBS, bit_set<LIMBS>::high_bit, false>::mprotate_left(r, numlimbs);
    }

}